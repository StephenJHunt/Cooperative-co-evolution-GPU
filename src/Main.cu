#include "hip/hip_runtime.h"

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

//include other files
#include "test.h"
#include "random.h"
//#include "environment.h"
#include "predatorprey.h"
//#include "network.h"
#include "feedforward.h"
#include "population.h"

#ifndef nPreds
#define nPreds = 3
#endif
#ifndef nHidden
#define nHidden = 15
#endif
//globals
Neuron* bestTeam;
int bestGene;
Population* subPops;
Population** predSubPops;
Gridworld world;
int catches;

//input params with default values
bool sim = false;
int hidden = 10;
int numIndivs = 100;
int numInputs = 2;
int numOutputs = 5;
int burstGens = 10;
int maxGens = 100000;
int goalFitness = 100000;
int numPreds = 3;
int trialsPerEval = 9;

struct tempState{
	int* PredatorX;
	int* PredatorY;
	int PreyX;
	int PreyY;
};

//struct aTeam{//moved to feedForward.h
//	int numOutputs;
//	int numInputs;
//	double act1[15];
//	Neuron t1[15];
//	double act2[15];
//	Neuron t2[15];
//	double act3[15];
//	Neuron t3[15];
//	int fitness;
//	int numHidden;
//	int catches;
//};

struct teamArr{
	aTeam team;
};


Population* init(int hid, int num, int genes){
	Population* pops = new Population[hid];
	for(int i = 0; i < hid; i++){
		Population* p = newPopulation(num, genes);
		createIndividuals(p);
		pops[i] = *p;
	}
	return pops;
}

int h_calculateDistance(Gridworld* h_world, int h_predX, int h_predY, int h_preyX, int h_preyY){
	double h_xDist = 0;
	double h_yDist = 0;

	h_xDist = abs((double)(h_predX-h_preyX));
	if(h_xDist > double(h_world->length/2)){
		h_xDist = double(h_world->length) - h_xDist;
	}

	h_yDist = abs((double)(h_predY-h_preyY));
		if(h_yDist > double(h_world->height/2)){
			h_yDist = double(h_world->height) - h_yDist;
		}
	return int(h_xDist + h_yDist);
}

__device__ int calculateDistance(Gridworld* world, int predX, int predY, int preyX ,int preyY){
	double xDist = 0;
	double yDist = 0;

	xDist = abs((double)(predX-preyX));
	if(xDist > double(world->length/2)){
		xDist = double(world->length) - xDist;
	}

	yDist = abs((double)(predY-preyY));
	if(yDist > double(world->height/2)){
		yDist = double(world->height) - yDist;
	}
	return int(xDist + yDist);
}

__global__ void kernelAssignFitness(int fitness, Neuron** hiddenUnits){
	int index = threadIdx.x + blockIdx.x * blockDim.x;

//    for(int i=index;i<numHidden;i++){
//		Neuron* n = hiddenUnits[i];
		hiddenUnits[index]->Fitness = fitness;
		hiddenUnits[index]->Trials++;
//		hiddenUnits[i] = n;
//    }
}

/*
feedForward* evaluate(PredatorPrey e, feedForward* team, int numTeams){
	catches = 0;
	int total_fitness = 0;
	int SMs;
	int deviceID;
	hipGetDevice(&deviceID);
	hipDeviceGetAttribute(&SMs, hipDeviceAttributeMultiprocessorCount, deviceID);
	int threadsPerBlock = 256;
	int blocks = 32 * SMs;

	int PreyPositions[2][9] = {{16, 50, 82, 82, 82, 16, 50, 50, 82},{50, 50, 50, 82, 16, 50, 16, 82, 50}};

	for(int l = 0;l < trialsPerEval;l++){//parallel?
		int fitness =0;
		int steps = 0;
		int maxSteps = 150;
		int avg_init_dist = 0;
		int avg_final_dist = 0;

		//do these before with hipMallocManaged
		int inplen = getTotalInputs(team[0]);
		int outlen = getTotalOutputs(team[0]);
		double* input = new double[inplen];
		double* output = new double[outlen];
		State state;

		setPreyPosition(e, PreyPositions[0][l], PreyPositions[1][l]);
		State* statepntr = getState(e);
		Gridworld* worldpntr = getWorld(e);
		state = *statepntr;
		world = *worldpntr;

		int nearestDist = 0;
		int nearestPred = 0;
		int currentDist = 0;

		for(int p = 0 ; p < numPreds; p++){
			avg_init_dist = avg_init_dist + calculateDistance(state.PredatorX[p], state.PredatorY[p], state.PreyX, state.PreyY);
		}
		avg_init_dist = avg_init_dist/numPreds;

		while(!Caught(e) && steps < maxSteps){//paralellise so that always runs maxSteps?
			for(int p=0; p < numPreds;p++){
				currentDist = calculateDistance(state.PredatorX[p], state.PredatorY[p], state.PreyX, state.PreyY);
				if(currentDist<nearestDist){
					nearestDist = currentDist;
					nearestPred = p;
				}
			}

			PerformPreyAction(e, nearestPred);

			for(int pred = 0; pred < numTeams;pred++){
				input[0] = double(e.state->PreyX);
				input[1] = double(e.state->PreyY);
				delete[] output;
				output = new double[outlen];//reset output in between?
				double* out = Activate(team[pred], input, inplen, output);
				PerformPredatorAction(e, pred, out, team[pred].NumOutputs);
//				printf("\n");
			}
			State* ts = getState(e);
			state = *ts;
			steps++;
//			delete[] input;
//			delete[] output;
///*
			//output state
			for(int pred = 0;pred < numPreds;pred++){
				printf("Predator %d, %d\n", state.PredatorX[pred], state.PredatorY[pred]);
			}
			printf("prey %d, %d \n", state.PreyX, state.PreyY);
//

		}

		if(Caught(e)){
			if(sim == true){
				printf("Simulation Complete\n");
				printf("Predator at position %d, %d caught the prey at position %d, %d after %d steps", state.PredatorX[nearestPred], state.PredatorY[nearestPred], state.PreyX, state.PreyY, steps);
			}
		}

		for(int p = 0; p < numPreds;p++){
			avg_final_dist = avg_final_dist + calculateDistance(state.PredatorX[p], state.PredatorY[p], state.PreyX, state.PreyY);
		}
		avg_final_dist = avg_final_dist/numPreds;

		if(!Caught(e)){
			fitness = (avg_init_dist - avg_final_dist);// /10
		}else{
			fitness = (200 - avg_final_dist)/10;
			catches++;
		}
		total_fitness = total_fitness + fitness;
	}

	for(int pred = 0; pred < numTeams;pred++){
		team[pred].Fitness = (total_fitness); // /trialsPerEval
		team[pred].Catches = catches;
		Neuron** d_neurons;
		// <<<blocks, threadsPerBlock>>>
		int numBytes = team[pred].numHidden * sizeof(team[pred].HiddenUnits[0]);
		//case 1
//		hipMalloc(&d_neurons, numBytes);//optimise to only take neuron fitness and trials not whole struct
//		hipMemcpy(team[pred].HiddenUnits, d_neurons, numBytes, hipMemcpyHostToDevice);
//		kernelAssignFitness<<<1, team[pred].numHidden>>>(total_fitness, d_neurons);
//		hipDeviceSynchronize();
//		hipMemcpy(team[pred].HiddenUnits, d_neurons, numBytes, hipMemcpyDeviceToHost);
		//case 2
		kernelAssignFitness<<<1, team[pred].numHidden>>>(total_fitness, team[pred].HiddenUnits);
		hipDeviceSynchronize();
		for(int i = 0; i<team[pred].numHidden;i++){
			Neuron* n = team[pred].HiddenUnits[i];
			n->Fitness = team[pred].Fitness;
			n->Trials++;
			team[pred].HiddenUnits[i] = n;
		}
	}
	return team;

}
*/

teamArr* h_eval(Gridworld* h_worldpntr, teamArr* h_teams, int h_numPreds, double* h_input, double* h_output, int h_inplen, int h_outlen, int h_trialsPerEval, bool h_sim, int h_numTrials){
	State* h_statepntr = new State();
	h_reset(h_statepntr, h_numPreds);

	int h_catches = 0;
	int h_totalfitness = 0;

	int h_PreyPositions[2][9] = {{16, 50, 82, 82, 82, 16, 50, 50, 82},{50, 50, 50, 82, 16, 50, 16, 82, 50}};

	for(int i=0;i<h_trialsPerEval;i++){
		int h_fitness = 0;
		int h_steps = 0;
		int h_maxSteps = 150;
		int h_avg_init_dist = 0;
		int h_avg_final_dist = 0;

		State h_state;
		Gridworld h_world;

		h_setPreyPosition(h_statepntr, h_PreyPositions[0][i], h_PreyPositions[1][i]);
		h_state = *h_statepntr;
		h_world = *h_worldpntr;

		int h_nearestDist = 100;
		int h_nearestPred = 0;
		int h_currentDist = 0;


	}
	return h_teams;
}

__global__ void runEvaluationsParallel(Gridworld* worldpntr, teamArr* d_teams, int numPreds, double* input, double* output, int inplen, int outlen, int trialsPerEval, bool sim, int numTrials){
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	State* statepntr = new State();
	kernelReset(statepntr, numPreds);
//
	for(int i = index;i < numTrials;i+= stride){
		int catches = 0;
		int total_fitness = 0;
//
		int PreyPositions[2][9] = {{16, 50, 82, 82, 82, 16, 50, 50, 82},{50, 50, 50, 82, 16, 50, 16, 82, 50}};
//
		for(int l = 0;l < trialsPerEval;l++){//parallel?
			int fitness =0;
			int steps = 0;
			int maxSteps = 150;
			int avg_init_dist = 0;
			int avg_final_dist = 0;

			State state;
			Gridworld world;

			setPreyPosition(statepntr, PreyPositions[0][l], PreyPositions[1][l]);//use state instead of PredatorPrey?
			state = *statepntr;
			world = *worldpntr;

			int nearestDist = 100;//so that closest pred changes
			int nearestPred = 0;
			int currentDist = 0;

			for(int p = 0 ; p < numPreds; p++){
				avg_init_dist = avg_init_dist + calculateDistance(worldpntr, state.PredatorX[p], state.PredatorY[p], state.PreyX, state.PreyY);
			}
			avg_init_dist = avg_init_dist/numPreds;

			while(!Caught(statepntr) && steps < maxSteps){//paralellise so that always runs maxSteps?
				for(int p=0; p < numPreds;p++){
					currentDist = calculateDistance(worldpntr, state.PredatorX[p], state.PredatorY[p], state.PreyX, state.PreyY);
					if(currentDist<nearestDist){
						nearestDist = currentDist;
						nearestPred = p;
					}
				}

				PerformPreyAction(statepntr, worldpntr, nearestPred);

				for(int pred = 0; pred < numPreds;pred++){
					input[0] = double(statepntr->PreyX);
					input[1] = double(statepntr->PreyY);
					delete[] output;
					output = new double[outlen];
					double* out = Activate(&d_teams[i].team, input, inplen, output);
					PerformPredatorAction(statepntr, worldpntr, pred, out, d_teams[i].team.numOutputs);//change to use state?
				}
				steps++;
			}
			if(Caught(statepntr)){
				if(sim == true){
					printf("Simulation Complete\n");
					printf("Predator at position %d, %d caught the prey at position %d, %d after %d steps", statepntr->PredatorX[nearestPred], statepntr->PredatorY[nearestPred], statepntr->PreyX, statepntr->PreyY, steps);
				}
			}

			for(int p = 0; p < numPreds;p++){
				avg_final_dist = avg_final_dist + calculateDistance(worldpntr, statepntr->PredatorX[p], statepntr->PredatorY[p], statepntr->PreyX, statepntr->PreyY);
			}
			avg_final_dist = avg_final_dist/numPreds;

			if(!Caught(statepntr)){
				fitness = (avg_init_dist - avg_final_dist);// /10
			}else{
				fitness = (200 - avg_final_dist)/10;
				catches++;
			}
			total_fitness = total_fitness + fitness;
		}

		d_teams[i].team.fitness = total_fitness; // /trialsPerEval
		d_teams[i].team.catches = catches;

		for(int i2 = 0; i2<d_teams[i].team.numHidden;i2++){
			Neuron n1 = d_teams[i].team.t1[i2];
			Neuron n2 = d_teams[i].team.t2[i2];
			Neuron n3 = d_teams[i].team.t3[i2];
			n1.Fitness = d_teams[i].team.fitness;
			n2.Fitness = d_teams[i].team.fitness;
			n3.Fitness = d_teams[i].team.fitness;
			n1.Trials++;
			n2.Trials++;
			n3.Trials++;
			d_teams[i].team.t1[i2] = n1;
			d_teams[i].team.t2[i2] = n2;
			d_teams[i].team.t3[i2] = n3;
		}
	}
}

void CHECK(hipError_t err){
	if(err){
		printf("Error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
	}
}


__global__ void testKernel(teamArr* teams, double* input, int inplen){
	int index = threadIdx.x;
//	input = new double[inplen];
	input[0] = 1.0;
//	double test = teams[index].team.t1[0].Weight[0];
//	teams[index].team.fitness = index+1;
	for(int i2 = 0; i2<teams[index].team.numHidden;i2++){
		Neuron n1 = teams[index].team.t1[i2];
		Neuron n2 = teams[index].team.t2[i2];
		Neuron n3 = teams[index].team.t3[i2];
		n1.Fitness = teams[index].team.fitness;
		n2.Fitness = teams[index].team.fitness;
		n3.Fitness = teams[index].team.fitness;
		n1.Trials++;
		n2.Trials++;
		n3.Trials++;
		teams[index].team.t1[i2] = n1;
		teams[index].team.t2[i2] = n2;
		teams[index].team.t3[i2] = n3;
		}
//	printf("Index %d Weight %d\n", index, test);
}

int main(int argc, char **argv)
{
	//testing values
	numInputs = 2;
	hidden = 15;
	numOutputs = 5;
	numIndivs = 100;//540
	maxGens = 100;
	goalFitness = 100;
	numPreds = 3;//6
	burstGens = 2;


	//TODO: parse input

	//simulation values
	bool stagnated;
	double mutationRate = 0.4;
	int bestFitness = 0;
	int generations = 0;
	stagnated = false;
	bool teamfound = false;
	int numTrials = 10 * numIndivs;

	//GPU values
	int SMs;
	int deviceID;
	hipGetDevice(&deviceID);
	hipDeviceGetAttribute(&SMs, hipDeviceAttributeMultiprocessorCount, deviceID);
	int threadsPerBlock = 256;
	int blocks = 32 * SMs;

	predSubPops = new Population*[numPreds];
	//initialisation of subpopulations
	for(int p = 0;p<numPreds;p++){
		feedForward* ff = newFeedForward(numInputs, hidden, numOutputs, false);
		Population* subpops = init(hidden, numIndivs, ff->GeneSize);
		predSubPops[p] = subpops;
	}

	teamArr* teams;
	teamArr* d_teams;

	//run simulation
	while(generations < maxGens && catches < numTrials){//run contents of this loop in parallel
		int numTeamBytes = numTrials * sizeof(aTeam);
		CHECK(hipMalloc(&d_teams, numTeamBytes));
		teams = (teamArr*)malloc(numTeamBytes);
		catches = 0;
		feedForward* ff = newFeedForward(numInputs, hidden, numOutputs, false);
		for(int t = 0; t < numTrials;t++){
			for(int p = 0;p<numPreds;p++){
				ff[p] = Create(ff[p], predSubPops[p], hidden);
			}
			aTeam tm;
			for(int i = 0;i<hidden;i++){
				tm.act1[i] = ff[0].Activation[i];
				tm.act2[i] = ff[1].Activation[i];
				tm.act3[i] = ff[2].Activation[i];
				tm.t1[i] = ff[0].HiddenUnits[i];
				tm.t2[i] = ff[1].HiddenUnits[i];
				tm.t3[i] = ff[2].HiddenUnits[i];
			}
			tm.catches = ff->Catches;
			tm.fitness = ff->Fitness;
			tm.numHidden = ff->numHidden;
			tm.numInputs = ff->NumInputs;
			tm.numOutputs = ff->NumOutputs;
			teams[t].team = tm;
		}
		CHECK(hipMemcpy(d_teams, teams, numTeamBytes, hipMemcpyHostToDevice));//State is causing the issue

		PredatorPrey* h_pp;
//		State* d_state;
		Gridworld* d_world;
//		CHECK(hipMalloc(&d_state, sizeof(State)));
		CHECK(hipMalloc(&d_world, sizeof(Gridworld)));
		h_pp = newPredatorPrey(numPreds);
		reset(h_pp, numPreds);
//		CHECK(hipMemcpy(d_state, h_pp->state, sizeof(State), hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_world, h_pp->world,  sizeof(Gridworld), hipMemcpyHostToDevice));
		//setup for kernel evaluation
		int inplen = (teams[0].team.numInputs);
		int outlen = (teams[0].team.numOutputs);
		double* d_input;
		double* h_input;
		CHECK(hipMalloc(&d_input, inplen * sizeof(double)));
		h_input = (double*)malloc(inplen * sizeof(double));
		double* d_output;
		double* h_output;
		CHECK(hipMalloc(&d_output, outlen * sizeof(double)));
		h_output = (double*)malloc(outlen * sizeof(double));

		//evaluate teams
//		testKernel<<<1, 100>>>(d_teams, d_input, inplen);
		// blocks, threadsPerBlock
//		runEvaluationsParallel<<<blocks, threadsPerBlock>>>(d_world, d_teams, numPreds, d_input, d_output, inplen, outlen, trialsPerEval, sim, numTrials);
//		feedForward* t = evaluate(*pp, team, numPreds);
		CHECK(hipPeekAtLastError());
//		hipDeviceSynchronize();
		//send memory back
		CHECK(hipMemcpy(teams, d_teams, numTeamBytes, hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(h_output, d_output, outlen * sizeof(double), hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(h_input, d_input, inplen * sizeof(double), hipMemcpyDeviceToHost));

		//assign team scores
		//TODO: loop through all teams
		for(int n = 0; n < numTrials;n++){
			catches = catches + (teams[n].team.catches);
			if(bestFitness == 0 && !teamfound){
				bestFitness = (teams[n].team.fitness);
			}

			//keep track of the best performing team
			if((teams[n].team.fitness) > bestFitness){
				bestFitness = (teams[n].team.fitness);
				bestGene = teams[n].team.numInputs + teams[n].team.numOutputs;
				bestTeam = teams[n].team.t1;
				//tag best team neurons
//				for(int i = 0;i<numPreds;i++){
//					Tag(bestTeam[0]);
//				}
			}
			//if this is the first run, take the team as the baseline best team
			if(!teamfound){
				teamfound = true;
				bestFitness = (teams[n].team.fitness);
				bestGene = teams[n].team.numInputs + teams[n].team.numOutputs;
				bestTeam = teams[n].team.t1;
			}
		}

		printf("Generation %d, best fitness is %d, catches is %d\n", generations+1, bestFitness, catches);

		//check for stagnation and burst mutate if stagnated
		if(generations%burstGens == 0 && generations != 0){
			//burst mutate
			stagnated = true;

			for(int pred = 0; pred < numPreds; pred++){
				Population* predPop = predSubPops[pred];
				for(int i = 0; i< hidden;i++){
					Population subpop = predPop[i];
					for(int n = 0; n< numIndivs;n++){
						Neuron indiv = subpop.Individuals[n];
						Neuron* hid = bestTeam;
						subpop.Individuals[n] = perturb(indiv, hid[i], bestGene);
					}
				}
			}
		}
		//sort by fitness, mate upper part and mutate populations if not stagnated
		if(!stagnated){
			for(int i = 0 ;i<numPreds;i++){
				for(int j = 0;j<hidden;j++){
					predSubPops[i][j] = sortNeurons(predSubPops[i][j]);
					predSubPops[i][j] = mate(predSubPops[i][j]);
					predSubPops[i][j] = mutate(predSubPops[i][j], mutationRate);
				}
			}
		}
		stagnated = false;
		generations++;
		CHECK(hipFree(d_teams));
		free(teams);
//		CHECK(hipFree(d_state));
		CHECK(hipFree(d_world));
		CHECK(hipFree(d_input));
		CHECK(hipFree(d_output));
		free(h_input);
		free(h_output);

	}
}

