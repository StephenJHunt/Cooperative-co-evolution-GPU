////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* Template project which demonstrates the basics on how to setup a project
* example application.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

//include other file
#include "test.h"
#include "random.h"


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char **argv)
{
	char* test = "hello";//this is a string now
	testSigmoid();
	//printf(test);
	//printf("\n");
    //printf("Hello World!\n");
    //CUDAHello<<<1,10>>>();
    hipDeviceReset();
}

